#include <stdio.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <time.h> 


void MatrixInit(float *M, int layers, int height, int width, bool init_zero) {
    srand(40); // Initialisation du générateur de nombres aléatoires

    for (int i = 0; i < layers; i++) {
        for (int j = 0; j < height; j++) {
            for (int k = 0; k < width; k++) {
                int index = i * height * width + j * width + k; // Calcul de l'index correct
                if (init_zero) {
                    M[index] = 0; // Initialisation à 0
                } else {
                    M[index] = (float)(rand()/(float)(RAND_MAX));    
                }
            }
        }
    }
}


void MatrixPrint(float *M, int width, int height, int depth) {
    // 'depth' est le nombre de matrices (6 dans ce cas),
    // 'width' et 'height' sont les dimensions de chaque matrice (14x14).

    for (int d = 0; d < depth; d++) {
        printf("Matrix %d:\n", d + 1);
        for (int i = 0; i < width; i++) {
            for (int j = 0; j < height; j++) {
                int index = d * (width * height) + i * height + j;
                printf("%f ", M[index]);
            }
            printf("\n");
        }
        printf("\n");
    }
}


void Matrix_kernel_init(float *M, int n, int p,int l){
    
    for(int i = 0; i < n; i++){
        for(int j = 0; j < p; j++){
            for(int k = 0; k < l; k++){
                printf("%f ", M[i*p + j*l + k]);
            }
            
        }
        printf("\n");
    }
}

__global__ void cudaMatrixMult_dot(float *M1, float *M2, float *Mout, int n){

    int i = blockDim.x * blockIdx.x + threadIdx.x; 
    
    *(Mout+i) = *(M1+i)*(*(M2+i));
}

__global__ void cudaConvolution2d(float *input, float *kernels, float *output, int input_size, int kernel_size, int n_kernels) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; //32
    int j = blockIdx.y * blockDim.y + threadIdx.y; //32
    int k = blockIdx.z; // 6

    int n_points = input_size - (kernel_size - 1); //28

    if (i < n_points && j < n_points && k < n_kernels) {
        float sum = 0.0;

        // Calcul des indices pour le noyau et l'output
        int kernelStartIndex = k * kernel_size * kernel_size;
        int outputIndex = k * n_points * n_points + i * n_points + j;

        // Boucle sur le noyau
        for (int ki = 0; ki < kernel_size; ki++) {
            for (int kj = 0; kj < kernel_size; kj++) {
                int inputIndex = (i + ki) * input_size + (j + kj);
                int kernelIndex = kernelStartIndex + ki * kernel_size + kj;
                sum += (input[inputIndex]) * (kernels[kernelIndex]);
            }
        }

        output[outputIndex] = sum;
    }
}

__global__ void cudaApplyTanh(float *input, float *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        output[idx] = tanhf(input[idx]); // Appliquer tanh à chaque élément
    }
}


__global__ void cudaAveragePooling(float *input, float *output, int inputWidth, int inputHeight, int inputDepth) {
    int outputWidth = inputWidth / 2; // 14
    int outputHeight = inputHeight / 2; // 14

    int i = blockIdx.x * blockDim.x + threadIdx.x; // 28
    int j = blockIdx.y * blockDim.y + threadIdx.y; // 28
    int k = blockIdx.z; // 6

    if (i < outputWidth && j < outputHeight && k < inputDepth) {
        float sum = 0.0;

        // Indices pour la fenêtre 2x2 dans l'image d'entrée
        for (int ki = 0; ki < 2; ki++) {
            for (int kj = 0; kj < 2; kj++) {
                int inputIndex = k * (inputWidth * inputHeight) + (2 * i + ki) * inputWidth + (2 * j + kj);
                sum += input[inputIndex];
            }
        }

        float average = sum / 4.0; // Calcul de la moyenne
        int outputIndex = k * (outputWidth * outputHeight) + i * outputWidth + j;
        output[outputIndex] = average;
    }
}

__global__ void denseLayer(float *input, float *weights, float *biases, float *output, int inputSize, int outputSize) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < outputSize) {
        float sum = 0.0;
        for (int i = 0; i < inputSize; i++) {
            sum += weights[row * inputSize + i] * input[i];
        }
        output[row] = sum + biases[row];
    }
}

int main(int argc, char *argv[]){

    float *raw_data,*C1_data,*C1_data_tanh,*S1_data,*C1_kernel;
    
    raw_data = (float*) malloc(sizeof(float)*32*32);
    C1_data = (float*) malloc(sizeof(float)*28*28*6);
    C1_data_tanh = (float*) malloc(sizeof(float)*28*28*6);
    S1_data = (float*) malloc(sizeof(float)*14*14*6);
    C1_kernel = (float*) malloc(sizeof(float)*5*5*6);



    //initializations
    MatrixInit(raw_data,32,32,1,false);
    MatrixInit(C1_kernel,5,5,6,false);

    float *raw_data_d,*C1_data_d,*C1_data_tanh_d,*S1_data_d,*C1_kernel_d;
    hipMalloc((void**)&raw_data_d, sizeof(float)*32*32);
    hipMalloc((void**)&C1_data_d, sizeof(float)*28*28*6);
    hipMalloc((void**)&C1_data_tanh_d, sizeof(float)*28*28*6);
    hipMalloc((void**)&S1_data_d, sizeof(float)*14*14*6);
    hipMalloc((void**)&C1_kernel_d, sizeof(float)*5*5*6);


    dim3 dimBlock(6, 6); // Taille du bloc (ajustez en fonction des performances et des limitations du matériel)
    dim3 dimBlocktanh(256);
    dim3 dimGridconv((28 + dimBlock.x - 1) / dimBlock.x, (28 + dimBlock.y - 1) / dimBlock.y, 6);
    dim3 dimGridpool((14 + dimBlock.x - 1) / dimBlock.x, (14 + dimBlock.y - 1) / dimBlock.y, 6);
    dim3 dimGridtanh((6*28*28 + dimBlock.x - 1) / dimBlock.x);

    hipMemcpy(raw_data_d,raw_data,sizeof(float)*32*32,hipMemcpyHostToDevice);
    hipMemcpy(C1_kernel_d,C1_kernel,sizeof(float)*5*5*6,hipMemcpyHostToDevice);


    hipEvent_t start_GPU, stop_GPU;
    hipEventCreate(&start_GPU);
    hipEventCreate(&stop_GPU);
    hipEventRecord(start_GPU, 0);
    cudaConvolution2d<<<dimGridconv, dimBlock>>>(raw_data_d, C1_kernel_d, C1_data_d, 32, 5, 6);
    cudaApplyTanh<<<dimGridtanh, dimBlocktanh>>>(C1_data_d, C1_data_tanh_d, 6*28*28);
    cudaAveragePooling<<<dimGridpool, dimBlock>>>(C1_data_tanh_d, S1_data_d, 28, 28, 6);
    hipEventRecord(stop_GPU, 0);
    hipEventSynchronize(stop_GPU);
    float elapsedTime_GPU;
    hipEventElapsedTime(&elapsedTime_GPU, start_GPU, stop_GPU);
    hipEventDestroy(start_GPU);
    hipEventDestroy(stop_GPU);
    printf("GPU conv: %f ms\n", elapsedTime_GPU);

    hipMemcpy(C1_data_tanh,C1_data_tanh_d,sizeof(float)*28*28*6,hipMemcpyDeviceToHost);
    hipMemcpy(S1_data,S1_data_d,sizeof(float)*14*14*6,hipMemcpyDeviceToHost);

    //MatrixPrint(C1_kernel,5,5,6);
    //MatrixPrint(raw_data,32,32,1);
    //MatrixPrint(C1_data,28,28,6);
    MatrixPrint(C1_data_tanh,28,28,6);
    //MatrixPrint(S1_data,14,14,6);

    printf("GPU conv: %f ms\n", elapsedTime_GPU);

    hipFree(raw_data_d);
    hipFree(C1_data_d);
    hipFree(C1_data_tanh_d);
    hipFree(S1_data_d);
    hipFree(C1_kernel_d);
    free(raw_data);
    free(C1_data);
    free(C1_data_tanh);
    free(S1_data);
    free(C1_kernel);
}
